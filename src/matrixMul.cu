#include "hip/hip_runtime.h"
#include <iostream>
#include <matrix.cu>
#include <chrono>
#define TIME std::chrono::steady_clock::time_point

TIME getTime(){
    return std::chrono::steady_clock::now();
}


template<typename T>
__global__ void cudaMatrixAdd(Matrix<T>* a, Matrix<T>*  b, Matrix<T>* c){
    int x = blockIdx.x*blockDim.x + threadIdx.x;
    int y = blockIdx.y*blockDim.y + threadIdx.y;



    if(c->isValid(x,y)){
        c->setValue(x, y, a->getValue(x,y) + b->getValue(x,y));
    }
} 
template<typename T> 
void cpuMatrixAdd(Matrix<T>* a, Matrix<T>*  b, Matrix<T>* c){
    for(int i = 0; i<c->getN();i++){
        for(int j = 0;j<c->getM();j++){
            c->setValue(i, j, a->getValue(i,j) + b->getValue(i,j));
        }
    }

}

template<typename T> 
bool verify(Matrix<T>* a, Matrix<T>*  b, Matrix<T>* c){
    for(int i = 0; i<c->getN();i++){
        for(int j = 0;j<c->getM();j++){
            if( c->getValue(i,j) !=  a->getValue(i,j) + b->getValue(i,j)){
                return false;
            }
        }
    }
    return true;
}
template<typename T>
Matrix<T>* cudaAllocateMatrix(Matrix<T> *matrix){
    Matrix<T>* d_matrix;
    hipError_t cudaErrorMalloc = hipMalloc((Matrix<T> **)&d_matrix, sizeof(*d_matrix));


    if (cudaErrorMalloc != hipSuccess) {
        fprintf(stderr, "CudaMallocFailed failed!");
     
    }
    hipError_t hipError_t = hipMemcpy(d_matrix, matrix, sizeof(*d_matrix), hipMemcpyHostToDevice);
    if (hipError_t != hipSuccess) {
        fprintf(stderr, "CudaMemoryMove failed!");
  
    }
  
    return d_matrix;
}

void MatrixAdd(int n, int m){
    Matrix<int> *a, *b, *c;
    a = new Matrix<int>(n,m);
    b = new Matrix<int>(n,m);
    c = new Matrix<int>(n,m);

    a->fillWithRandomInt(1000);
    b->fillWithRandomInt(1000);

    Matrix<int> *d_a, *d_b, *d_c;

    a->moveToGPU();
    b->moveToGPU();
    c->moveToGPU();


    d_a = cudaAllocateMatrix(a);
    d_b = cudaAllocateMatrix(b);
    d_c = cudaAllocateMatrix(c);

    int threadsPerBlock = 32;
    dim3 block(threadsPerBlock, threadsPerBlock);
    dim3 grid(a->getN()/threadsPerBlock + 1, a->getM()/threadsPerBlock + 1);



    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start);
    cudaMatrixAdd<<<grid, block>>>(d_a, d_b, d_c);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float GPUmilliseconds = 0;
    hipEventElapsedTime(&GPUmilliseconds, start, stop);

    a->moveToCPU();
    b->moveToCPU();
    c->moveToCPU();

    std::cout << "GPU Completed Addition in " << GPUmilliseconds << " [ms]" << std::endl;
    if(!verify(a, b, c)){
        std::cout << "Matrix Addition Verification Has Failed" << std::endl;
    }

    TIME begin = getTime();
    cpuMatrixAdd(a, b, c);
    TIME end = getTime();
    std::cout << "CPU Completed Addition in " << (std::chrono::duration_cast<std::chrono::milliseconds>(end-begin).count() )<< " [ms]" << std::endl;
    
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    delete a;
    delete b;
    delete c;
}
int main(){
    int n, m;
    std::cin >> n >> m;
    MatrixAdd(n,m);

    return 0;
}